#include "include/fastpy/cuda.hpp"

// Allocate CUDA memory
void* cuda_malloc(size_t size) {
    void* ptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    return ptr;
}

// Free CUDA memory
void cuda_free(void* ptr) {
    if (ptr != nullptr) {
        CUDA_CHECK(hipFree(ptr));
    }
}

// Copy data from host to device
void cuda_memcpy_host_to_device(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

// Copy data from device to host
void cuda_memcpy_device_to_host(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

// Copy data from device to device
void cuda_memcpy_device_to_device(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
}
